#include "hip/hip_runtime.h"
#include <limits>

#include <ATen/ATen.h>
#include <ATen/WrapDimUtils.h>
#include <ATen/LegacyTHFunctionsCUDA.h>
#include <ATen/core/Array.h>
#include <ATen/cuda/cub.cuh>
#include <ATen/cuda/detail/KernelUtils.h>

namespace at { namespace native {

bool should_use_th_sort(const Tensor &self, int64_t dim) {
  int64_t ndim = self.dim();
  dim = maybe_wrap_dim(dim, ndim);
  int64_t nsort = self.sizes()[dim];
  int64_t threshold;
  if (self.scalar_type() == kLong || self.scalar_type() == kDouble) {
    threshold = 1024;
  } else {
    threshold = 2048;
  }
  return nsort <= threshold;
}

std::vector<int64_t> infer_dense_strides_dim_last(const Tensor & self, int64_t dim);

// If the dim being sorted is smaller than 2048/1024, then we will use the
// implementation THC. Otherwise we use cub's segmented sort
std::tuple<Tensor &,Tensor &> sort_out_stable_cuda(const Tensor & self, c10::optional<bool> stable, int64_t dim, bool descending, Tensor & values, Tensor & indices) {
  if (should_use_th_sort(self, dim)) {
    return legacy::cuda::_th_sort_out_stable(self, stable, dim, descending, values, indices);
  }
  // this algorithm is always stable
  TORCH_INTERNAL_ASSERT(stable.has_value(), "sort_out(): c10::optional<bool> for stable has to have value.");
  bool is_non_overlapping_and_dense = self.is_non_overlapping_and_dense();
  int64_t numel = self.numel();
  int64_t ndim = self.dim();
  dim = maybe_wrap_dim(dim, ndim);
  int64_t nsort = self.sizes()[dim];

  TORCH_CHECK(nsort <= std::numeric_limits<int>::max(),
    "The dimension being sorted can not have more than INT_MAX elsments.");

  if (ndim == 0) {
    if (!values.defined()) {
      values = self.clone();
    } else {
      values.resize_as_(self);
      values.copy_(self);
    }
    if (!indices.defined()) {
      indices = at::zeros({}, self.options().dtype(kLong));
    } else {
      indices.resize_as_(self);
      indices.zero_();
    }
    return std::forward_as_tuple(values, indices);
  }

  Tensor self_;
  if (is_non_overlapping_and_dense && self.stride(dim) == 1) {
    self_ = self;
  } else {
    auto new_strides_unsort = infer_dense_strides_dim_last(self, dim);
    self_ = at::empty_strided(self.sizes(), new_strides_unsort, self.options());
    self_.copy_(self);
  }

  Tensor values_tmp, indices_tmp;
  void *values_ptr_;
  int64_t *indices_ptr;
  if (!values.defined()) {
    if (is_non_overlapping_and_dense) {
      values = at::empty_strided(self.sizes(), self.strides(), self.options());
    } else {
      auto strides = at::infer_dense_strides(self.sizes(), self.strides());
      values = at::empty_strided(self.sizes(), strides, self.options());
    }
  } else {
    TORCH_CHECK(self_.scalar_type() == values.scalar_type(),
      "Unexpected dtype for values, expect ", self_.scalar_type(), ", got ", values.scalar_type());
    values.resize_as_(self);
  }
  if (values.strides() != self_.strides()) {
    values_tmp = at::empty_strided(self_.sizes(), self_.strides(), self_.options());
    values_ptr_ = values_tmp.data_ptr();
  } else {
    values_ptr_ = values.data_ptr();
  }

  if (!indices.defined()) {
    if (is_non_overlapping_and_dense) {
      indices = at::empty_strided(self.sizes(), self.strides(), self.options().dtype(kLong));
    } else {
      auto strides = at::infer_dense_strides(self.sizes(), self.strides());
      indices = at::empty_strided(self.sizes(), strides, self.options().dtype(kLong));
    }
  } else {
    TORCH_CHECK(kLong == indices.scalar_type(),
      "Unexpected dtype for values, expect torch.long, got ", indices.scalar_type());
    indices.resize_as_(self);
  }
  if (indices.strides() != self_.strides()) {
    indices_tmp = at::empty_strided(self_.sizes(), self_.strides(), self_.options().dtype(kLong));
    indices_ptr = indices_tmp.data_ptr<int64_t>();
  } else {
    indices_ptr = indices.data_ptr<int64_t>();
  }

  if (numel == 0) {
    return std::forward_as_tuple(values, indices);
  }

  int64_t numel_or_intmax = std::min(numel, static_cast<int64_t>(std::numeric_limits<int>::max()));
  int64_t nbatch = (numel_or_intmax / nsort) * nsort;

  AT_DISPATCH_ALL_TYPES_AND2(kBool, kHalf, self_.scalar_type(), "sort", [&]{
    const scalar_t *self_ptr = self_.data_ptr<scalar_t>();
    auto values_ptr = reinterpret_cast<scalar_t *>(values_ptr_);
    int64_t remaining = numel;
    while (remaining > 0) {
      int64_t n = std::min(remaining, nbatch);
      int64_t nsegments = n / nsort;

      auto int_options = indices.options().dtype(kInt);
      auto offset_begins = at::arange(0, n, nsort, int_options);
      auto offset_ends = at::arange(nsort, n + nsort, nsort, int_options);
      auto reverse_indices = at::arange(nsort, indices.options()).view({1, nsort}).expand({nsegments, nsort}).contiguous();

      at::cuda::hipcub::segmented_sort_pairs(self_ptr, values_ptr,
        reverse_indices.data_ptr<int64_t>(), indices_ptr, n, nsegments,
        offset_begins.data_ptr<int>(), offset_ends.data_ptr<int>(), descending);

      remaining -= n;
      self_ptr += n;
      values_ptr += n;
      indices_ptr += n;
    }
  });

  if (values_tmp.defined()) {
    values.copy_(values_tmp);
  }
  if (indices_tmp.defined()) {
    indices.copy_(indices_tmp);
  }
  return std::forward_as_tuple(values, indices);
}

// If the dim being sorted is smaller than 2048/1024, then we will use the
// implementation THC. Otherwise we use cub's segmented sort
std::tuple<Tensor &,Tensor &> sort_out_cuda(const Tensor & self, int64_t dim, bool descending, Tensor & values, Tensor & indices) {
  if (should_use_th_sort(self, dim)) {
    return legacy::cuda::_th_sort_out(self, dim, descending, values, indices);
  }
  return sort_out_stable_cuda(self, /*stable=*/false, dim, descending, values, indices);
}

// If the dim being sorted is smaller than 2048/1024, then we will use the
// implementation THC. Otherwise we use cub's segmented sort
std::tuple<Tensor,Tensor> sort_stable_cuda(const Tensor & self, c10::optional<bool> stable, int64_t dim, bool descending) {
  if (should_use_th_sort(self, dim)) {
    return legacy::cuda::_th_sort_stable(self, stable, dim, descending);
  }
  Tensor values, indices;
  return sort_out_stable_cuda(self, stable, dim, descending, values, indices);
}

// If the dim being sorted is smaller than 2048/1024, then we will use the
// implementation THC. Otherwise we use cub's segmented sort
std::tuple<Tensor,Tensor> sort_cuda(const Tensor & self, int64_t dim, bool descending) {  int64_t threshold;
  if (should_use_th_sort(self, dim)) {
    return legacy::cuda::_th_sort(self, dim, descending);
  }
  return sort_stable_cuda(self, /*stable=*/false, dim, descending);
}

}}  // namespace at::native
